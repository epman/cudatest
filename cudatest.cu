
#include <time.h>
#include <stdlib.h>
#include <stdint.h>

#include <iostream>

// Required to include CUDA vector types
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#define LOOP_COUNT	10000000

__global__
void gpu_loopTest()
{
  long tmp=0;
  for (int i=0; i<LOOP_COUNT; i++) {
    tmp++;
  }
}

void cpu_loopTest()
{
  long tmp=0;
  for (int i=0; i<LOOP_COUNT; i++) {
    tmp++;
  }
}


int main() 
{
  
  std::cout << "CUDA test" << std::endl;
  struct timespec t0, t1; 

  // Loop
  clock_gettime(CLOCK_MONOTONIC, &t0); 
  cpu_loopTest();
  clock_gettime(CLOCK_MONOTONIC, &t1); 
  uint64_t dt = 1000000000L * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  std::cout << "CPU Loop test. Time: " << dt << " ns" << std::endl;

  clock_gettime(CLOCK_MONOTONIC, &t0); 
  dim3 dimBlock( 1, 1 );
  dim3 dimGrid( 1, 1 );
  gpu_loopTest <<< dimGrid, dimBlock >>> ();
  clock_gettime(CLOCK_MONOTONIC, &t1); 
  dt = 1000000000L * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  std::cout << "GPU Loop test. Time: " << dt << " ns" << std::endl;

  return 0;
}
