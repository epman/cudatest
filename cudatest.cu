
#include <time.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>

#include <iostream>

// Required to include CUDA vector types
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#define REPETITIONS	1000

#define NS_PER_SECOND	1000000000L


#define VECTOR_SIZE	1000*1024
// CUDA kernel. Each thread takes care of one element of c
// From https://www.olcf.ornl.gov/tutorials/cuda-vector-addition
__global__ 
void kernel_vector_sum(float *dst, float *v, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        dst[id] += v[id];
}


void cpu_vector_sum(float *dst, float *v, int n)
{
  for (int i=0; i<n; i++) 
        dst[n] += v[n];
}


void vectorSumTest()
{
  struct timespec t0, t1; 
  uint64_t dt;
  float *dst_cpu = new float[VECTOR_SIZE];
  float *v_cpu = new float[VECTOR_SIZE];
  std::cout << "CPU Vector Sum test. ";
  dt = 0;
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    cpu_vector_sum(dst_cpu, v_cpu, VECTOR_SIZE);
    clock_gettime(CLOCK_MONOTONIC, &t1); 
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time: " << ((double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;

  float *dst_gpu;
  float *v_gpu;
  int numCudaThreads = 1;
  hipMalloc(&dst_gpu, VECTOR_SIZE*sizeof(float));
  hipMalloc(&v_gpu, VECTOR_SIZE*sizeof(float));
  dt = 0;
  int blockSize = 1024;
  int gridSize = (int)ceil((float)numCudaThreads/blockSize);
  std::cout << "GPU Vector Sum test. ";
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    kernel_vector_sum <<< gridSize, blockSize >>> (dst_gpu, v_gpu, VECTOR_SIZE);
    clock_gettime(CLOCK_MONOTONIC, &t1);     
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time for "<< numCudaThreads << " threads: " << ( (double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;
  
  
  numCudaThreads = 64;
  dt = 0;
  blockSize = 1024;
  gridSize = (int)ceil((float)numCudaThreads/blockSize);
  std::cout << "GPU Vector Sum test. ";
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    kernel_vector_sum <<< gridSize, blockSize >>> (dst_gpu, v_gpu, VECTOR_SIZE);
    clock_gettime(CLOCK_MONOTONIC, &t1);     
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time for "<< numCudaThreads << " threads: " << ( (double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;

  numCudaThreads = 128;
  dt = 0;
  blockSize = 1024;
  gridSize = (int)ceil((float)numCudaThreads/blockSize);
  std::cout << "GPU Vector Sum test. ";
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    kernel_vector_sum <<< gridSize, blockSize >>> (dst_gpu, v_gpu, VECTOR_SIZE);
    clock_gettime(CLOCK_MONOTONIC, &t1);     
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time for "<< numCudaThreads << " threads: " << ( (double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;
  
  numCudaThreads = 128;
  dt = 0;
  blockSize = 1024;
  gridSize = (int)ceil((float)numCudaThreads/blockSize);
  std::cout << "GPU Vector Sum test. ";
  const int vsize = VECTOR_SIZE*sizeof(float);
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    hipMemcpy( dst_gpu, dst_cpu, vsize, hipMemcpyHostToDevice );
    hipMemcpy( v_gpu, v_cpu, vsize, hipMemcpyHostToDevice );   
    kernel_vector_sum <<< gridSize, blockSize >>> (dst_gpu, v_gpu, VECTOR_SIZE);
    hipMemcpy( dst_cpu, dst_gpu, vsize, hipMemcpyDeviceToHost );
    clock_gettime(CLOCK_MONOTONIC, &t1);     
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time for "<< numCudaThreads << " threads (with memcpy): " << ( (double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;

  hipFree(&dst_gpu);
  hipFree(&v_gpu);
  delete[] v_cpu;
  delete[] dst_cpu;
  
}



#define LOOP_COUNT	10000000


__global__
void gpu_loopTest()
{
  long tmp=0;
  for (int i=0; i<LOOP_COUNT; i++) {
    tmp++;
  }
}

void cpu_loopTest()
{
  long tmp=0;
  for (int i=0; i<LOOP_COUNT; i++) {
    tmp++;
  }
}

// Loop
void loopTest()
{
  struct timespec t0, t1; 
  uint64_t dt;
  std::cout << "CPU Loop test. ";
  dt = 0;
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    cpu_loopTest();
    clock_gettime(CLOCK_MONOTONIC, &t1); 
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time: " << ((double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;

  dt = 0;
  dim3 dimBlock( 1, 1 );
  dim3 dimGrid( 1, 1 );
  std::cout << "GPU Loop test. ";
  for (int r=0; r<REPETITIONS; r++) 
  {
    clock_gettime(CLOCK_MONOTONIC, &t0); 
    gpu_loopTest <<< dimGrid, dimBlock >>> ();
    clock_gettime(CLOCK_MONOTONIC, &t1);     
    dt += NS_PER_SECOND * (t1.tv_sec - t0.tv_sec) + t1.tv_nsec - t0.tv_nsec;
  }
  std::cout << "Time: " << ( (double)dt/(double)(REPETITIONS*NS_PER_SECOND)) << " s (" << (dt/REPETITIONS) << " ns)" << std::endl;
}


int main() 
{
  std::cout.imbue( std::locale("") );
  std::cout << "CUDA test" << std::endl;
  std::cout << "Averages for " << REPETITIONS << " repetitions." << std::endl;
  
  loopTest();
  vectorSumTest();

  return 0;
}
